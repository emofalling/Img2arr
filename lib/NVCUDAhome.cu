#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// compile: nvcc NVCUDAhome.cu -shared -o NVCUDAhome.dll -O3 -lcuda -lcudart (-Xcompiler "/source-charset:utf-8")(for Windows)

#if defined(_WIN32) || defined(_WIN64)
#define SHARED __declspec(dllexport)
#else
#define SHARED __attributes__((visibility("default")))
#endif

#include <stdbool.h>

typedef unsigned int uint;

#if defined(__cplusplus)
extern "C" {
#endif

inline SHARED hipError_t api_cuInit(uint Flags){
    return hipInit(Flags);
}

inline SHARED hipError_t api_cuDeviceGetCount(int *count){
    return hipGetDeviceCount(count);
}

inline SHARED hipError_t api_cuDeviceGet(hipDevice_t *device, uint ordinal){
    return hipDeviceGet(device, ordinal);
}

inline SHARED hipError_t api_cuDeviceGetName(char *name, int len, hipDevice_t dev){
    return hipDeviceGetName(name, len, dev);
}

inline SHARED hipError_t api_cuDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev){
    return hipDeviceGetAttribute(pi, attrib, dev);
}

// 初始化上下文
inline SHARED hipError_t api_cuCtxCreate(hipCtx_t *pctx, uint flags, hipDevice_t dev){
    return hipCtxCreate(pctx, flags, dev);
}

// 销毁上下文
inline SHARED hipError_t api_cuCtxDestroy(hipCtx_t ctx){
    return hipCtxDestroy(ctx);
}


/*
module: 模块句柄
image: 二进制文件数据（ptx cubin fatbin均可）
numOptions: 选项数量
*/
inline SHARED hipError_t api_cuModuleLoadDataEx(hipModule_t *module, const void *image, uint numOptions, hipJitOption *options, void **optionValues){
    return hipModuleLoadDataEx(module, image, numOptions, options, optionValues);
}

// 卸载模块
inline SHARED hipError_t api_cuModuleUnload(hipModule_t module){
    return hipModuleUnload(module);
}

// 取函数
inline SHARED hipError_t api_cuModuleGetFunction(hipFunction_t *hfunc, hipModule_t hmod, const char *name){
    return hipModuleGetFunction(hfunc, hmod, name);
}

// 分配内存
inline SHARED hipError_t api_cudaMalloc(void **dptr, size_t size){
    return hipMalloc(dptr, size);
}

// 分配内存（主机）。相比malloc，这个函数分配的内存是不分页的，能够有效地使用hipMemcpyAsync复制
inline SHARED hipError_t api_cudaMallocHost(void **ptr, size_t size){
    return hipHostMalloc(ptr, size);
}

// 清理内存
inline SHARED hipError_t api_cudaFree(void *dptr){
    return hipFree(dptr);
}

// 启动！
inline SHARED hipError_t api_cuLaunchKernel(hipFunction_t f, uint gridDimX, uint gridDimY, uint gridDimZ, uint blockDimX, uint blockDimY, uint blockDimZ, uint sharedMemBytes, hipStream_t hStream, void **kernelParams, void **extra){
    return hipModuleLaunchKernel(f, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, hStream, kernelParams, extra);
}

// 等待整个上下文完成
inline SHARED hipError_t api_cuCtxSynchronize(){
    return hipCtxSynchronize();
}

// 等待某个流完成
inline SHARED hipError_t api_cuStreamSynchronize(hipStream_t stream){
    return hipStreamSynchronize(stream);
}

// 串行复制内存。kind指定了复制的方向
SHARED hipError_t api_cudaMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind) {
    return hipMemcpy(dst, src, count, kind);
}

// 异步复制内存。kind指定了复制的方向
SHARED hipError_t api_cudaMemcpyAsync(void* dst, const void* src, size_t count, hipMemcpyKind kind, hipStream_t stream) {
    return hipMemcpyAsync(dst, src, count, kind, stream);
}

// 创建流
SHARED hipError_t api_cudaStreamCreate(hipStream_t *pStream) {
    return hipStreamCreate(pStream);
}

// 销毁流
SHARED hipError_t api_cudaStreamDestroy(hipStream_t stream) {
    return hipStreamDestroy(stream);
}

SHARED const char* api_cudaGetErrorString(hipError_t error) {
    return hipGetErrorString(error);
}

#if defined(__cplusplus)
}
#endif